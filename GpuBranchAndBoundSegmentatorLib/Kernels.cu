#include "hip/hip_runtime.h"
#include "CudaMath.h"
#include "Kernels.h"

__device__ float DistanceSqrToObjectPenalty(float distanceSqr, float edgeWidthSqr, float backgroundDistanceCoeff)
{
	return distanceSqr;
}

__device__ float DistanceSqrToBackgroundPenalty(float distanceSqr, float edgeWidthSqr, float backgroundDistanceCoeff)
{
	return max(edgeWidthSqr * (1 + backgroundDistanceCoeff) - backgroundDistanceCoeff * distanceSqr, 0.f);
}

// Clockwise order from bottom left (min) corner assumed
__device__ float2 ProjectToConstraints(float2 point, float2 corners[4])
{
	return trunc(point, corners[0], corners[2]);
}

#define BLOCK_DIM 16
#define INFINITY 1e+20f

__device__ __constant__ float2 EdgeConvexHull[8];
__device__ __constant__ float2 Corners1[4];
__device__ __constant__ float2 Corners2[4];

__global__ void CalcMinPenaltiesForEdgeKernel(
    int2 imageSize,
    int edgeConvexHullSize,
	float backgroundDistanceCoeff,
    float2 minMaxWidthSqr,
    float *objectPenalties,
	float *backgroundPenalties)
{
    int2 pointInt;
    pointInt.x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    pointInt.y = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (pointInt.x >= imageSize.x || pointInt.y >= imageSize.y)
        return;
    int index = pointInt.x + pointInt.y * imageSize.x;
    float2 point = make_float2(pointInt.x, pointInt.y);

    float minDistanceSqr = INFINITY;
	float maxDistanceSqr = 0;

    if (PointInConvexHull(point, EdgeConvexHull, edgeConvexHullSize))
        minDistanceSqr = 0;
	
	for (int i = 0; i < 4; ++i)
	{
		for (int j = 0; j < 4; ++j)
		{
			float distanceSqr = DistanceToSegmentSqr(point, Corners1[i], Corners2[j]);
			minDistanceSqr = min(minDistanceSqr, distanceSqr);
			maxDistanceSqr = max(maxDistanceSqr, distanceSqr);
		}
	}
	
	float2 projection1 = ProjectToConstraints(point, Corners1);
	for (int i = 0; i < 4; ++i)
		minDistanceSqr = min(minDistanceSqr, DistanceToSegmentSqr(point, projection1, Corners2[i]));

	float2 projection2 = ProjectToConstraints(point, Corners2);
	for (int i = 0; i < 4; ++i)
		minDistanceSqr = min(minDistanceSqr, DistanceToSegmentSqr(point, Corners1[i], projection2));

	float minObjectPenalty = DistanceSqrToObjectPenalty(minDistanceSqr, minMaxWidthSqr.y, backgroundDistanceCoeff);
	float minBackgroundPenalty = DistanceSqrToBackgroundPenalty(maxDistanceSqr, minMaxWidthSqr.x, backgroundDistanceCoeff);

    objectPenalties[index] = min(objectPenalties[index], minObjectPenalty);
	backgroundPenalties[index] = max(backgroundPenalties[index], minBackgroundPenalty);
}

void CalculateShapeUnaryTerms(
	int edgeCount,
	float2 **convexHulls,
	int *convexHullSizes,
	float2 **corners1,
	float2 **corners2,
	float2 *edgeWidthLimits,
	float backgroundDistanceCoeff,
    int imageWidth,
    int imageHeight,
    float *objectPenalties,
    float *backgroundPenalties)
{
	int totalImageSize = imageWidth * imageHeight;
    
    // Prepare GPU storage
    float *objectPenaltiesGPU;
	float *backgroundPenaltiesGPU;
	int totalStorageSize = totalImageSize * sizeof(float);
    hipMalloc((void**) &objectPenaltiesGPU, totalStorageSize);
	hipMalloc((void**) &backgroundPenaltiesGPU, totalStorageSize);
    
    // Cleanup storage
	for (int i = 0; i < totalImageSize; ++i)
    {
		objectPenalties[i] = INFINITY;
		backgroundPenalties[i] = -INFINITY;
    }
    hipMemcpy(objectPenaltiesGPU, objectPenalties, totalStorageSize, hipMemcpyHostToDevice);
	hipMemcpy(backgroundPenaltiesGPU, backgroundPenalties, totalStorageSize, hipMemcpyHostToDevice);

    for (int i = 0; i < edgeCount; ++i)
    {
        // Setup convex hull for the current edge
        hipMemcpyToSymbol(HIP_SYMBOL("EdgeConvexHull"), convexHulls[i], convexHullSizes[i] * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("Corners1"), corners1[i], 4 * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("Corners2"), corners2[i], 4 * sizeof(float2));
        
        // Prepare GPU grid
        dim3 blockDim(BLOCK_DIM, BLOCK_DIM, 1);
        dim3 gridDim((imageWidth + blockDim.x - 1) / blockDim.x, (imageHeight + blockDim.y - 1) / blockDim.y, 1);
        
        // Run kernel to update penalty storage
        CalcMinPenaltiesForEdgeKernel<<<gridDim, blockDim, 0>>>(
            make_int2(imageWidth, imageHeight),
            convexHullSizes[i],
			backgroundDistanceCoeff,
            make_float2(edgeWidthLimits[i].x * edgeWidthLimits[i].x, edgeWidthLimits[i].y * edgeWidthLimits[i].y),
            objectPenaltiesGPU,
			backgroundPenaltiesGPU);

		hipDeviceSynchronize();
    }

	// Save results
    hipMemcpy(objectPenalties, objectPenaltiesGPU, totalStorageSize, hipMemcpyDeviceToHost);
	hipMemcpy(backgroundPenalties, backgroundPenaltiesGPU, totalStorageSize, hipMemcpyDeviceToHost);
    
    hipFree(objectPenaltiesGPU);
	hipFree(backgroundPenaltiesGPU);
}