#include "hip/hip_runtime.h"
#include "CudaMath.h"
#include "Kernels.h"

__device__ float DistanceToObjectPenalty(float distance, float cutoff)
{
    return -log_inf(exp(-cutoff * distance * distance));
}

__device__ float DistanceToBackgroundPenalty(float distance, float cutoff)
{
    return -log_inf(1 - exp(-cutoff * distance * distance));
}

__device__ float2 GetClosestPoint(float2 point, float2 corners[4])
{
    // Clockwise order from bottom left (min) corner assumed
    float2 min = corners[0];
    float2 max = corners[2];
    
    if (point.x >= min.x && point.x <= max.x)
    {
        if (point.y <= min.y)
            return make_float2(point.x, min.y);
        if (point.y >= max.y)
            return make_float2(point.x, max.y);
    }

    if (point.y >= min.y && point.y <= max.y)
    {
        if (point.x <= min.x)
            return make_float2(min.x, point.y);
        if (point.x >= max.x)
            return make_float2(max.x, point.y);
    }

    // Just because we have to return some point
    return min;
}

#define BLOCK_DIM 16
#define INFINITY 1e+20f

__device__ __constant__ float2 VertexCorners1[4];
__device__ __constant__ float2 VertexCorners2[4];
__device__ __constant__ float2 EdgeConvexHull[8];

__global__ void CalcMinObjectPenaltyKernel(
    int2 imageSize,
    int edgeConvexHullSize,
    float2 maxRadii,
    float distanceCutoff,
    float *minObjectPenalties)
{
    int2 pointInt;
    pointInt.x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    pointInt.y = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (pointInt.x >= imageSize.x || pointInt.y >= imageSize.y)
        return;
    int index = pointInt.x + pointInt.y * imageSize.x;
    float2 point = make_float2(pointInt.x, pointInt.y);

    float distance = INFINITY;
    if (PointInConvexHull(point, EdgeConvexHull, edgeConvexHullSize))
    {
        distance = 0;
    }
    else
    {
		for (int corner1 = 0; corner1 < 4; ++corner1)
            for (int corner2 = 0; corner2 < 4; ++corner2)
            {
                float distanceToEdge = DistanceToPulleyArea(
                    point,
                    VertexCorners1[corner1],
                    maxRadii.x,
                    VertexCorners2[corner2],
                    maxRadii.y);
                distance = min(distance, distanceToEdge);
            }

        float2 closestPoint1 = GetClosestPoint(point, VertexCorners1);
        float2 closestPoint2 = GetClosestPoint(point, VertexCorners2);

        for (int corner = 0; corner < 4; ++corner)
        {
            float distanceToEdge1 = DistanceToPulleyArea(
                point,
                closestPoint1,
                maxRadii.x,
                VertexCorners2[corner],
                maxRadii.y);
            distance = min(distance, distanceToEdge1);

            float distanceToEdge2 = DistanceToPulleyArea(
                point,
                VertexCorners1[corner], 
                maxRadii.x,
                closestPoint2,
                maxRadii.y);
            distance = min(distance, distanceToEdge2);
        }

        float distanceBetweenClosestPoints = DistanceToPulleyArea(
                point,
                closestPoint1,
                maxRadii.x,
                closestPoint2,
                maxRadii.y);
        distance = min(distance, distanceBetweenClosestPoints);
    }

    float penalty = DistanceToObjectPenalty(distance, distanceCutoff);
    minObjectPenalties[index] = min(minObjectPenalties[index], penalty);
}

__global__ void CalcMaxBackgroundPenaltyKernel(
    int2 imageSize,
    int edgeConvexHullSize,
    float2 minRadii,
    float distanceCutoff,
    float *maxBackgroundPenalties)
{
    int2 pointInt;
    pointInt.x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    pointInt.y = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (pointInt.x >= imageSize.x || pointInt.y >= imageSize.y)
        return;
    int index = pointInt.x + pointInt.y * imageSize.x;
    float2 point = make_float2(pointInt.x, pointInt.y);

    float distance = 0;
	for (int corner1 = 0; corner1 < 4; ++corner1)
        for (int corner2 = 0; corner2 < 4; ++corner2)
        {
            float distanceToEdge = DistanceToPulleyArea(
                point,
                VertexCorners1[corner1],
                minRadii.x,
                VertexCorners2[corner2],
                minRadii.y);
            distance = max(distance, distanceToEdge);
        }

    float penalty = DistanceToBackgroundPenalty(distance, distanceCutoff);
    maxBackgroundPenalties[index] = max(maxBackgroundPenalties[index], penalty);
}

void CalculateShapeUnaryTerms(
	int edgeCount,
	float2 **corners1,
	float2 **corners2,
	float2 **convexHulls,
	int *convexHullSizes,
	float2 *minRadii,
	float2 *maxRadii,
    int imageWidth,
    int imageHeight,
    float distanceCutoff,
    float *objectPenalties,
    float *backgroundPenalties)
{
	int totalImageSize = imageWidth * imageHeight;
    
    // Prepare GPU storage
    float *objectPenaltiesGPU;
	float *backgroundPenaltiesGPU;
	int totalStorageSize = totalImageSize * sizeof(float);
    hipMalloc((void**) &objectPenaltiesGPU, totalStorageSize);
	hipMalloc((void**) &backgroundPenaltiesGPU, totalStorageSize);
    
    // Cleanup storage
	for (int i = 0; i < totalImageSize; ++i)
    {
		objectPenalties[i] = INFINITY;
		backgroundPenalties[i] = -INFINITY;
    }
    hipMemcpy(objectPenaltiesGPU, objectPenalties, totalStorageSize, hipMemcpyHostToDevice);
	hipMemcpy(backgroundPenaltiesGPU, backgroundPenalties, totalStorageSize, hipMemcpyHostToDevice);

    // Calculate min object penalty
    for (int i = 0; i < edgeCount; ++i)
    {
        // Prepare corners and convex hull
        hipMemcpyToSymbol(HIP_SYMBOL("VertexCorners1"), corners1[i], 4 * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("VertexCorners2"), corners2[i], 4 * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("EdgeConvexHull"), convexHulls[i], convexHullSizes[i] * sizeof(float2));
        
        // Prepare GPU grid
        dim3 blockDim(BLOCK_DIM, BLOCK_DIM, 1);
        dim3 gridDim((imageWidth + blockDim.x - 1) / blockDim.x, (imageHeight + blockDim.y - 1) / blockDim.y, 1);
        
        // Run kernel to update penalty storage
        CalcMinObjectPenaltyKernel<<<gridDim, blockDim, 0>>>(
            make_int2(imageWidth, imageHeight),
            convexHullSizes[i],
            maxRadii[i],
            distanceCutoff,
            objectPenaltiesGPU);
		CalcMaxBackgroundPenaltyKernel<<<gridDim, blockDim, 0>>>(
            make_int2(imageWidth, imageHeight),
            convexHullSizes[i],
            minRadii[i],
            distanceCutoff,
            backgroundPenaltiesGPU);

		hipDeviceSynchronize();
    }

	// Save results
    hipMemcpy(objectPenalties, objectPenaltiesGPU, totalStorageSize, hipMemcpyDeviceToHost);
	hipMemcpy(backgroundPenalties, backgroundPenaltiesGPU, totalStorageSize, hipMemcpyDeviceToHost);
    
    hipFree(objectPenaltiesGPU);
	hipFree(backgroundPenaltiesGPU);
}