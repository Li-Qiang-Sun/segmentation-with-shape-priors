#include "hip/hip_runtime.h"
#include "CudaMath.h"
#include "Kernels.h"

#define BLOCK_DIM 16
#define INFINITY 1e+20f
#define LOG_2 0.69314718

__device__ float DistanceSqrToObjectPenalty(float distanceSqr, float edgeWidthSqr)
{
	return 4 * LOG_2 * distanceSqr / (edgeWidthSqr + 1e-6);
}

__device__ float DistanceSqrToBackgroundPenalty(float distanceSqr, float edgeWidthSqr)
{
	return -log(1 + 1e-6 - exp(-DistanceSqrToObjectPenalty(distanceSqr, edgeWidthSqr)));
}

__device__ __constant__ float2 EdgeConvexHull[8];
__device__ __constant__ float2 Corners1[4]; 
__device__ __constant__ float2 Corners2[4];

__global__ void CalcMinPenaltiesForEdgeKernel(
    int2 imageSize,
    int edgeConvexHullSize,
    float2 minMaxWidthSqr,
    float *objectPenalties,
	float *backgroundPenalties)
{
    int2 pointInt;
    pointInt.x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    pointInt.y = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (pointInt.x >= imageSize.x || pointInt.y >= imageSize.y)
        return;
    int index = pointInt.x + pointInt.y * imageSize.x;
    float2 point = make_float2(pointInt.x, pointInt.y);

	float minDistanceSqr;
	if (PointInConvexHull(point, EdgeConvexHull, edgeConvexHullSize))
        minDistanceSqr = 0;
	else
	{
		minDistanceSqr = INFINITY;
		for (int i = 0; i < edgeConvexHullSize; ++i)
		{
			float distanceSqr = DistanceToSegmentSqr(point, EdgeConvexHull[i], EdgeConvexHull[(i + 1) % edgeConvexHullSize]);
			minDistanceSqr = min(minDistanceSqr, distanceSqr);
		}
	}
	
	float maxDistanceSqr = 0;
	for (int i = 0; i < 4; ++i)
	{
		for (int j = 0; j < 4; ++j)
		{
			float distanceSqr = DistanceToSegmentSqr(point, Corners1[i], Corners2[j]);
			maxDistanceSqr = max(maxDistanceSqr, distanceSqr);
		}
	}
	
	float minObjectPenalty = DistanceSqrToObjectPenalty(minDistanceSqr, minMaxWidthSqr.y);
	float minBackgroundPenalty = DistanceSqrToBackgroundPenalty(maxDistanceSqr, minMaxWidthSqr.x);

    objectPenalties[index] = min(objectPenalties[index], minObjectPenalty);
	backgroundPenalties[index] = max(backgroundPenalties[index], minBackgroundPenalty);
}

void CalculateShapeUnaryTerms(
	int edgeCount,
	float2 **convexHulls,
	int *convexHullSizes,
	float2 **corners1,
	float2 **corners2,
	float2 *edgeWidthLimits,
    int imageWidth,
    int imageHeight,
    float *objectPenalties,
    float *backgroundPenalties)
{
    for (int i = 0; i < edgeCount; ++i)
    {
        // Setup convex hull for the current edge
        hipMemcpyToSymbol(HIP_SYMBOL("EdgeConvexHull"), convexHulls[i], convexHullSizes[i] * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("Corners1"), corners1[i], 4 * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("Corners2"), corners2[i], 4 * sizeof(float2));
        
        // Prepare GPU grid
        dim3 blockDim(BLOCK_DIM, BLOCK_DIM, 1);
        dim3 gridDim((imageWidth + blockDim.x - 1) / blockDim.x, (imageHeight + blockDim.y - 1) / blockDim.y, 1);
        
        // Run kernel to update penalty storage
        CalcMinPenaltiesForEdgeKernel<<<gridDim, blockDim, 0>>>(
            make_int2(imageWidth, imageHeight),
            convexHullSizes[i],
            make_float2(edgeWidthLimits[i].x * edgeWidthLimits[i].x, edgeWidthLimits[i].y * edgeWidthLimits[i].y),
            objectPenalties,
			backgroundPenalties);

		hipDeviceSynchronize();
    }
}