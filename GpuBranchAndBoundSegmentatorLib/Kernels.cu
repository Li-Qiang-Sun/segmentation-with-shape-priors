#include "hip/hip_runtime.h"
#include "CudaMath.h"
#include "Kernels.h"

__device__ float DistanceSqrToObjectPenalty(float distanceSqr, float edgeWidthSqr, float backgroundDistanceCoeff)
{
	return distanceSqr;
}

__device__ float DistanceSqrToBackgroundPenalty(float distanceSqr, float edgeWidthSqr, float backgroundDistanceCoeff)
{
	return max(0.25 * edgeWidthSqr * (1 + backgroundDistanceCoeff) - backgroundDistanceCoeff * distanceSqr, 0.f);
}

#define BLOCK_DIM 16
#define INFINITY 1e+20f

__device__ __constant__ float2 EdgeConvexHull[8];
__device__ __constant__ float2 Corners1[4]; 
__device__ __constant__ float2 Corners2[4];

__global__ void CalcMinPenaltiesForEdgeKernel(
    int2 imageSize,
    int edgeConvexHullSize,
	float backgroundDistanceCoeff,
    float2 minMaxWidthSqr,
    float *objectPenalties,
	float *backgroundPenalties)
{
    int2 pointInt;
    pointInt.x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    pointInt.y = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (pointInt.x >= imageSize.x || pointInt.y >= imageSize.y)
        return;
    int index = pointInt.x + pointInt.y * imageSize.x;
    float2 point = make_float2(pointInt.x, pointInt.y);

	float minDistanceSqr;
	if (PointInConvexHull(point, EdgeConvexHull, edgeConvexHullSize))
        minDistanceSqr = 0;
	else
	{
		minDistanceSqr = INFINITY;
		for (int i = 0; i < edgeConvexHullSize; ++i)
		{
			float distanceSqr = DistanceToSegmentSqr(point, EdgeConvexHull[i], EdgeConvexHull[(i + 1) % edgeConvexHullSize]);
			minDistanceSqr = min(minDistanceSqr, distanceSqr);
		}
	}
	
	float maxDistanceSqr = 0;
	for (int i = 0; i < 4; ++i)
	{
		for (int j = 0; j < 4; ++j)
		{
			float distanceSqr = DistanceToSegmentSqr(point, Corners1[i], Corners2[j]);
			maxDistanceSqr = max(maxDistanceSqr, distanceSqr);
		}
	}
	
	float minObjectPenalty = DistanceSqrToObjectPenalty(minDistanceSqr, minMaxWidthSqr.y, backgroundDistanceCoeff);
	float minBackgroundPenalty = DistanceSqrToBackgroundPenalty(maxDistanceSqr, minMaxWidthSqr.x, backgroundDistanceCoeff);

    objectPenalties[index] = min(objectPenalties[index], minObjectPenalty);
	backgroundPenalties[index] = max(backgroundPenalties[index], minBackgroundPenalty);
}

void CalculateShapeUnaryTerms(
	int edgeCount,
	float2 **convexHulls,
	int *convexHullSizes,
	float2 **corners1,
	float2 **corners2,
	float2 *edgeWidthLimits,
	float backgroundDistanceCoeff,
    int imageWidth,
    int imageHeight,
    float *objectPenalties,
    float *backgroundPenalties)
{
    for (int i = 0; i < edgeCount; ++i)
    {
        // Setup convex hull for the current edge
        hipMemcpyToSymbol(HIP_SYMBOL("EdgeConvexHull"), convexHulls[i], convexHullSizes[i] * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("Corners1"), corners1[i], 4 * sizeof(float2));
		hipMemcpyToSymbol(HIP_SYMBOL("Corners2"), corners2[i], 4 * sizeof(float2));
        
        // Prepare GPU grid
        dim3 blockDim(BLOCK_DIM, BLOCK_DIM, 1);
        dim3 gridDim((imageWidth + blockDim.x - 1) / blockDim.x, (imageHeight + blockDim.y - 1) / blockDim.y, 1);
        
        // Run kernel to update penalty storage
        CalcMinPenaltiesForEdgeKernel<<<gridDim, blockDim, 0>>>(
            make_int2(imageWidth, imageHeight),
            convexHullSizes[i],
			backgroundDistanceCoeff,
            make_float2(edgeWidthLimits[i].x * edgeWidthLimits[i].x, edgeWidthLimits[i].y * edgeWidthLimits[i].y),
            objectPenalties,
			backgroundPenalties);

		hipDeviceSynchronize();
    }
}